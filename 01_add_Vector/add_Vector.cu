#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <utils.h>

using namespace std;

void addVectorCPU(float* a, float* b, float* c, const int length)
{
    for(int i=0; i<length; i++){
        c[i] = a[i] + b[i];
    }
}

__global__ void addVectorGPU(float* a, float* b, float* c, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N)
        c[i] = a[i] + b[i];
}

int main(int argc, char** argv)
{
    int dev = 0;
    hipSetDevice(dev);

    int nElems = 1<<24;
    int nBytes = sizeof(float) * nElems;
    float* a_host = (float*)malloc(nBytes);
    float* b_host = (float*)malloc(nBytes);
    float* c_host = (float*)malloc(nBytes);
    float* c_from_dev_host = (float*)malloc(nBytes);

    //初始化
    initialData(a_host, nElems);
    initialData(b_host, nElems);
    memset(c_host, 0, nBytes);
    memset(c_from_dev_host, 0, nBytes);

    double iStart, iElaps;

    //GPU
    //定义设备内存
    float *a_dev, *b_dev, *c_dev;
    CHECK(hipMalloc((float**)&a_dev, nBytes));
    CHECK(hipMalloc((float**)&b_dev, nBytes));
    CHECK(hipMalloc((float**)&c_dev, nBytes));

    CHECK(hipMemcpy(a_dev, a_host, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_dev, b_host, nBytes, hipMemcpyHostToDevice));

    dim3 block(512);
    dim3 grid((nElems-1)/block.x+1);
    iStart = cpuSecond();
    addVectorGPU<<<grid, block>>>(a_dev, b_dev, c_dev, nElems);
    iElaps = cpuSecond() - iStart;
    printf("<<<%d, %d>>>, Time elapsed %f sec\n", block.x, grid.x, iElaps);

    CHECK(hipMemcpy(c_from_dev_host, c_dev, nBytes, hipMemcpyDeviceToHost));

    //CPU 
    iStart = cpuSecond();
    addVectorCPU(a_host, b_host, c_host, nElems);
    iElaps = cpuSecond() - iStart;
    printf("Time elapsed %f sec\n", block.x, grid.x, iElaps);

    checkResult(c_host, c_from_dev_host, nElems);

    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);

    free(a_host);
    free(b_host);
    free(c_host);
    free(c_from_dev_host);

    return 0;
}